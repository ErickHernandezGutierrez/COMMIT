#include "hip/hip_runtime.h"
#include "gpumanager.cuh"

bool cudaCheck(hipError_t cudaStatus){
    return cudaStatus == hipSuccess;
}

void preprocessDataForGPU(uint32_t* data, int NUM_COMPARTMENTS, uint32_t* compartmentsPerBlock, uint32_t* offsetPerBlock, int NUM_BLOCKS){

    // fill arrays with zeros
    memset(compartmentsPerBlock, 0, NUM_BLOCKS * sizeof(uint32_t));
    memset(offsetPerBlock,       0, NUM_BLOCKS * sizeof(uint32_t));

    // count compartments per block
    for(int i = 0; i < NUM_COMPARTMENTS; i++)
        compartmentsPerBlock[data[i]]++;

    // calculate offset per block
    offsetPerBlock[0] = 0;
    for(int i = 1; i < NUM_BLOCKS; i++)
        offsetPerBlock[i] = offsetPerBlock[i-1] + compartmentsPerBlock[i-1];
}

/*
__dual__ segment::segment() {}

__dual__ segment::~segment() {}
//*/

CudaLinearOperator::CudaLinearOperator(
    uint32_t* voxelIC,
    uint32_t* fiberIC,
    uint16_t* orienIC,
    float*    lengthIC,
    float*    lutIC,

    uint32_t* voxelEC,
    uint16_t* orienEC,
    float*    lutEC,

    float*    lutISO,

    int nsegments,
    int nvoxels,      
    int nfibers,      
    int npeaks,       
    int norientations,
    int nsamples,     
    int ndiameters,   
    int nzeppelins,   
    int nballs)
{
    this->nvoxels = nvoxels;
    this->nfibers = nfibers;
    this->nrows = nvoxels * nsamples;
    this->ncols = nfibers*ndiameters + npeaks*nzeppelins + nvoxels*nballs;
    int size_lutic  = ndiameters*norientations*nsamples;
    int size_lutec  = nzeppelins*norientations*nsamples;
    int size_lutiso = nballs*nsamples;
    bool status;

    uint32_t* segmentsPerBlock = (uint32_t*) malloc(nvoxels*sizeof(uint32_t));
    uint32_t* offsetPerBlock   = (uint32_t*) malloc(nvoxels*sizeof(uint32_t));

    // copy constant values to GPU
    printf("\t* constant global values ... ");
    status = true;
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_VOXELS),       &nvoxels,       sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_FIBERS),       &nfibers,       sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_PEAKS),        &npeaks,        sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_ORIENTATIONS), &norientations, sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_SAMPLES),      &nsamples,      sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_DIAMETERS),    &ndiameters,    sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_ZEPPELINS),    &nzeppelins,    sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_BALLS),        &nballs,        sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_ROWS),         &nrows,         sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_COLS),         &ncols,         sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTIC),       &size_lutic,    sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTEC),       &size_lutec,    sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTISO),      &size_lutiso,   sizeof(int)) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");


    // alloc memory in GPU for vectors x and y
    printf("\t* memory for vectors x and y ... ");
    status = true;
    status = status && cudaCheck( hipMalloc((void**)&(this->x), ncols*sizeof(float64_t)) );
    status = status && cudaCheck( hipMalloc((void**)&(this->y), nrows*sizeof(float64_t)) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    // alloc GPU memory for segments
    printf("\t* memory for LUT (IC part) ... ");
    status = true;
    status = status && cudaCheck( hipMalloc((void**)&(this->lutIC), size_lutic*sizeof(float32_t)) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* copying LUT in GPU (IC part) ... ");
    status = true;
    status = status && cudaCheck( hipMemcpy(this->lutIC, lutIC, size_lutic*sizeof(float32_t), hipMemcpyHostToDevice) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* allocating memory for LUT in GPU (EC part) ... ");
    status = cudaCheck( hipMalloc((void**)&(this->lutEC), size_lutec*sizeof(float32_t)) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* copying LUT in GPU (EC part) ... ");
    status = cudaCheck( hipMemcpy(this->lutEC, lutEC, size_lutec*sizeof(float32_t), hipMemcpyHostToDevice) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* allocating memory for LUT in GPU (ISO part) ... ");
    status = cudaCheck( hipMalloc((void**)&(this->lutISO), size_lutiso*sizeof(float32_t)) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* copying LUT in GPU (ISO part) ... ");
    status = cudaCheck( hipMemcpy(this->lutISO, lutISO, size_lutiso*sizeof(float32_t), hipMemcpyHostToDevice) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* preprocessing data for GPU ... ");
    preprocessDataForGPU(voxelIC, nsegments, segmentsPerBlock, offsetPerBlock, nvoxels);
    printf("\n");

    printf("\t* fiber segments memory allocation ... ");
    status = true;
    status = status && cudaCheck( hipMalloc((void**)&(this->voxelIC),  nsegments*sizeof(uint32_t))  );
    status = status && cudaCheck( hipMalloc((void**)&(this->fiberIC),  nsegments*sizeof(uint32_t))  );
    status = status && cudaCheck( hipMalloc((void**)&(this->orienIC),  nsegments*sizeof(uint16_t))  );
    status = status && cudaCheck( hipMalloc((void**)&(this->lengthIC), nsegments*sizeof(float32_t)) );
    status = status && cudaCheck( hipMalloc((void**)&(this->segmentsPerBlockIC), nvoxels*sizeof(uint32_t)) );
    status = status && cudaCheck( hipMalloc((void**)&(this->offsetPerBlockIC),   nvoxels*sizeof(uint32_t)) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* transfering fiber segments ... ");
    status = true;
    status = status && cudaCheck( hipMemcpy(this->voxelIC,  voxelIC,  nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->fiberIC,  fiberIC,  nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->orienIC,  orienIC,  nsegments*sizeof(uint16_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->lengthIC, lengthIC, nsegments*sizeof(float32_t), hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->segmentsPerBlockIC, segmentsPerBlock, nvoxels*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->offsetPerBlockIC,   offsetPerBlock,   nvoxels*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    // ---------------------------------------- EC DATA ---------------------------------------- //
    printf("\t* allocating memory for operator A in GPU (EC part) ... ");
    status = true;
    status = status && cudaCheck( hipMalloc((void**)&(this->voxelEC),  npeaks*sizeof(uint32_t)) );
    status = status && cudaCheck( hipMalloc((void**)&(this->orienEC),  npeaks*sizeof(uint16_t)) );
    status = status && cudaCheck( hipMalloc((void**)&(this->segmentsPerBlockEC), nvoxels*sizeof(uint32_t))  );
    status = status && cudaCheck( hipMalloc((void**)&(this->offsetPerBlockEC),   nvoxels*sizeof(uint32_t))  );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* preprocessing EC data for GPU ... ");
    preprocessDataForGPU(voxelEC, npeaks, segmentsPerBlock, offsetPerBlock, nvoxels);
    printf("\n");

    printf("\t* copying operator A to GPU (EC part) ... ");
    status = true;
    status = status && cudaCheck( hipMemcpy(this->voxelEC,            voxelEC,              npeaks*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->orienEC,            orienEC,              npeaks*sizeof(uint16_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->segmentsPerBlockEC, segmentsPerBlock,     nvoxels*sizeof(uint32_t), hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->offsetPerBlockEC,   offsetPerBlock,       nvoxels*sizeof(uint32_t), hipMemcpyHostToDevice) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    free(segmentsPerBlock);
    free(offsetPerBlock);
}

CudaLinearOperator::~CudaLinearOperator(){
    hipFree(voxelIC);
    hipFree(fiberIC);
    hipFree(orienIC);
    hipFree(lengthIC);
    hipFree(lutIC);
    hipFree(segmentsPerBlockIC);
    hipFree(offsetPerBlockIC);
    
    hipFree(voxelEC);
    hipFree(orienEC);
    hipFree(lutEC);
    hipFree(segmentsPerBlockEC);
    hipFree(offsetPerBlockEC);

    hipFree(lutISO);

    hipFree(x);
    hipFree(y);
}

__global__ void multiply_Ax_ICpart(
    uint32_t*  voxelIDs,
    uint32_t*  fiberIDs,
    uint16_t*  orienIDs,
    float32_t* lengths,
    uint32_t*  segmentsPerBlock,
    uint32_t*  offsetPerBlock,
    float32_t* lut,
    float64_t* x,
    float64_t* y){

    __shared__ float64_t shmem[1024];

    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;
    uint32_t gid = threadIdx.x / 512;
    uint32_t sid = threadIdx.x - 512*gid;

    shmem[tid] = 0.0;

    if(sid >= NUM_SAMPLES) return;

    uint32_t offset = offsetPerBlock[bid] + (segmentsPerBlock[bid]/2)*gid;
    uint32_t nsegments = segmentsPerBlock[bid]/2 + (segmentsPerBlock[bid]%2)*gid;

    //segment_t* segment = segments + offset;
    uint32_t*  voxel  = voxelIDs + offset;
    uint32_t*  fiber  = fiberIDs + offset;
    uint16_t*  orien  = orienIDs + offset;
    float32_t* length = lengths  + offset;

    float64_t sum = 0.0;

    for(int i = 0; i < nsegments; i++){
        int offset_lut = (*orien)*NUM_SAMPLES + sid;

        float64_t aux = 0.0;
        for(int j = 0; j < NUM_DIAMETERS; j++){
            aux += (double)(lut[offset_lut + j*NUM_ORIENTATIONS*NUM_SAMPLES])*x[(*fiber) + j*NUM_FIBERS];
            //aux += tex1Dfetch(tex_lutIC, offset_lut + j*num_orientations*num_samples) * x[(*fiber) + j*num_fibers];
        }

        sum += aux * (*length);

        fiber++;
        orien++;
        length++;
    }

    shmem[tid] = sum;
    __syncthreads();

    if(tid < NUM_SAMPLES)
        y[(*voxel)*NUM_SAMPLES + sid] = sum + shmem[tid+512];
}

__global__ void multiply_Ax_ECpart(
    uint32_t*  voxelIDs,
    uint16_t*  orienIDs,
    uint32_t*  segmentsPerBlock,
    uint32_t*  offsetPerBlock,
    float32_t* lut,
    float64_t* x,
    float64_t* y)
{
    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;

    if(tid >= NUM_SAMPLES) return;

    uint32_t offset  = offsetPerBlock[bid];
    uint32_t nsegments = segmentsPerBlock[bid];

    //compartmentEC_t* excomp = excomps + offset;
    uint32_t* voxel = voxelIDs + offset;
    uint16_t* orien = orienIDs + offset;

    uint32_t target = NUM_FIBERS*NUM_DIAMETERS + offset;

    float64_t sum = 0.0;
    for(int i = 0; i < nsegments; i++){
        uint32_t offset_lut = (*orien)*NUM_SAMPLES + tid;

        for(int j = 0; j < NUM_ZEPPELINS; j++)
            sum += (double)(lut[lut_offset + j*NUM_ORIENTATIONS*NUM_SAMPLES])*x[target + j*NUM_PEAKS + i];
            //sum += tex1Dfetch(tex_lutEC, offset_lut + j*num_orientations*num_samples) * x[target + j*num_excomps + i];

        orien++;
    }

    y[(*voxel)*NUM_SAMPLES + tid] += sum;
}

__global__ void multiply_Ax_ISOpart(
    float32_t* lut,
    float64_t* x,
    float64_t* y)
{
    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;

    if(tid >= NUM_SAMPLES) return;

    uint32_t target = NUM_FIBERS*NUM_DIAMETERS + NUM_PEAKS*NUM_ZEPPELINS + bid;

    float64_t sum = 0.0;
    for(int j = 0; j < NUM_BALLS; j++)
        sum += (double)(lut[j*NUM_SAMPLES + tid])*x[target + j*NUM_VOXELS];
        //sum += (double)(tex1Dfetch(tex_lutISO, j*num_samples + tid))*x[target + j*num_voxels];
        

    y[bid*NUM_SAMPLES + tid] += sum;
}

void CudaLinearOperator::multiplyByX(float64_t* x, float64_t* y){

    // Copy vector x to the GPU
    hipMemcpy(this->x, x, ncols*sizeof(double), hipMemcpyHostToDevice);

    // Multiply IC part in the GPU
    multiply_Ax_ICpart<<<nvoxels, 1024>>>(voxelIC, fiberIC, orienIC, lengthIC, segmentsPerBlockIC, offsetPerBlockIC, lutIC, this->x, this->y);

    //cudaCheckKernel();

    // Multiply EC part in the GPU
    multiply_Ax_ECpart<<<nvoxels, 512>>>(voxelEC, orienEC, segmentsPerBlockEC, offsetPerBlockEC, lutEC, this->x, this->y);

    //cudaCheckKernel();

    // Multiply ISO part in the GPU
    multiply_Ax_ISOpart<<<nvoxels, 512>>>(lutISO, this->x, this->y);

    //cudaCheckKernel();

    // Copy back result to CPU
    hipMemcpy(y, this->y, nrows*sizeof(double), hipMemcpyDeviceToHost);
}

void CudaLinearOperator::multiplyByY(float64_t* y, float64_t* x){
        
    // Copy vector y to the GPU
    //cudaCheck( hipMemset(gpu_x, 0, NUM_COLS*sizeof(float64_t)) );
    //cudaCheck( hipMemcpy(gpu_x, x, NUM_COLS*sizeof(double), hipMemcpyHostToDevice) );
    //cudaCheck( hipMemcpy(gpu_y, y, NUM_ROWS*sizeof(double), hipMemcpyHostToDevice) );

    // Multiply IC part in the GPU
    //multiply_Aty_ICpart<<<NUM_FIBERS, 512>>>(gpu_voxelICt, gpu_fiberICt, gpu_orientICt, gpu_lengthICt, gpu_segmentsPerBlockICt, gpu_offsetPerBlockICt, gpu_lutIC, gpu_x, gpu_y);

    //cudaCheckKernel();//*/

    // Multiply EC part in the GPU
    //multiply_Aty_ECpart<<<NUM_VOXELS, 512>>>(gpu_voxelEC, gpu_orientEC, gpu_segmentsPerBlockEC, gpu_offsetPerBlockEC, gpu_lutEC, gpu_x, gpu_y);

    //cudaCheckKernel();

    // Multiply ISO part in the GPU
    //multiply_Aty_ISOpart<<<NUM_VOXELS, 512>>>(gpu_lutISO, gpu_x, gpu_y);

    //cudaCheckKernel();//*/

    // Copy back result to CPU
    //cudaCheck( hipMemcpy(x, gpu_x, NUM_COLS*sizeof(double), hipMemcpyDeviceToHost) ); 
        
    /*printf("\n\n VECTOR X EC PART:\n");
    for(int i = NUM_FIBERS*NUM_RESFUNCIC; i < NUM_FIBERS*NUM_RESFUNCIC+20; i++)
        printf("%lf ", x[i]);
    printf("\n\n");//*/
}
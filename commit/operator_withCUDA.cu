#include "hip/hip_runtime.h"
#include "operator_withCUDA.cuh"

CudaLinearOperator::CudaLinearOperator(
    // pointers to IC data in CPU memory
    uint32_t* voxelIC,
    uint32_t* fiberIC,
    uint16_t* orienIC,
    float*    lengthIC,
    float*    lutIC,
    // pointers to EC data in CPU memory
    uint32_t* voxelEC,
    uint16_t* orienEC,
    float*    lutEC,
    // pointer to ISO data in CPU memory
    float*    lutISO,
    // dataset constant values
    int nsegments,
    int nvoxels,      
    int nfibers,      
    int npeaks,       
    int norientations,
    int nsamples,     
    int ndiameters,   
    int nzeppelins,   
    int nballs,

    bool fcall)
{
    this->nsegments = nsegments;
    this->nvoxels   = nvoxels;
    this->nfibers   = nfibers;
    this->nrows     = nvoxels * nsamples;
    this->ncols     = nfibers*ndiameters + npeaks*nzeppelins + nvoxels*nballs;

    if (fcall) {
        printf("\t* configuring dictionary in CUDA GPU ... \n");

        int size_lutic  = ndiameters*norientations*nsamples;
        int size_lutec  = nzeppelins*norientations*nsamples;
        int size_lutiso = nballs*nsamples;

        size_t required_mem = 28*nsegments + 6.0*npeaks + 8.0*nfibers + 16.0*nvoxels + 4.0*(size_lutic + size_lutec + size_lutiso + this->nrows + this->ncols);
        checkCompatibility(required_mem);

        cudaStatus = true;

        cudaStatus = cudaStatus && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_VOXELS),       &nvoxels,       sizeof(int)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_FIBERS),       &nfibers,       sizeof(int)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_PEAKS),        &npeaks,        sizeof(int)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_ORIENTATIONS), &norientations, sizeof(int)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_SAMPLES),      &nsamples,      sizeof(int)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_DIAMETERS),    &ndiameters,    sizeof(int)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_ZEPPELINS),    &nzeppelins,    sizeof(int)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_BALLS),        &nballs,        sizeof(int)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_ROWS),         &nrows,         sizeof(int)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_COLS),         &ncols,         sizeof(int)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTIC),       &size_lutic,    sizeof(int)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTEC),       &size_lutec,    sizeof(int)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTISO),      &size_lutiso,   sizeof(int)) );

        // alloc memory in GPU for vectors x and y
        cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_x, ncols*sizeof(float64_t)) );
        cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_y, nrows*sizeof(float64_t)) );

        // setup LUTs
        cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_lutIC, size_lutic*sizeof(float32_t)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_lutIC, lutIC, size_lutic*sizeof(float32_t), hipMemcpyHostToDevice) );

        cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_lutEC,  size_lutec*sizeof(float32_t)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_lutEC, lutEC, size_lutec*sizeof(float32_t), hipMemcpyHostToDevice) );

        cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_lutISO, size_lutiso*sizeof(float32_t)) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_lutISO, lutISO, size_lutiso*sizeof(float32_t), hipMemcpyHostToDevice) );

        //if (gpu_voxelIC == NULL || gpu_fiberIC == NULL || gpu_orienIC == NULL || gpu_lengthIC == NULL) {
            cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_voxelIC,  nsegments*sizeof(uint32_t))  );
            cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_fiberIC,  nsegments*sizeof(uint32_t))  );
            cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_orienIC,  nsegments*sizeof(uint16_t))  );
            cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_lengthIC, nsegments*sizeof(float32_t)) );

            cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_voxelIC,  voxelIC,  nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice) );
            cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_fiberIC,  fiberIC,  nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice) );
            cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_orienIC,  orienIC,  nsegments*sizeof(uint16_t),  hipMemcpyHostToDevice) );
            cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_lengthIC, lengthIC, nsegments*sizeof(float32_t), hipMemcpyHostToDevice) );
        //}
        
        //if (gpu_segmentsPerBlockIC == NULL || gpu_offsetPerBlockIC == NULL) {
            uint32_t* segmentsPerBlock = (uint32_t*) malloc(nvoxels*sizeof(uint32_t));
            uint32_t* offsetPerBlock   = (uint32_t*) malloc(nvoxels*sizeof(uint32_t));

            preprocessDataForGPU(voxelIC, nsegments, segmentsPerBlock, offsetPerBlock, nvoxels);

            cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_segmentsPerBlockIC, nvoxels*sizeof(uint32_t)) );
            cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_offsetPerBlockIC,   nvoxels*sizeof(uint32_t)) );

            cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_segmentsPerBlockIC, segmentsPerBlock, nvoxels*sizeof(uint32_t),  hipMemcpyHostToDevice) );
            cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_offsetPerBlockIC,   offsetPerBlock,   nvoxels*sizeof(uint32_t),  hipMemcpyHostToDevice) );

            //free(segmentsPerBlock);
            //free(offsetPerBlock);
        //}

        //if (gpu_voxelEC == NULL || gpu_orienEC == NULL) {
            cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_voxelEC,  npeaks*sizeof(uint32_t)) );
            cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_orienEC,  npeaks*sizeof(uint16_t)) );

            cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_voxelEC, voxelEC, npeaks*sizeof(uint32_t), hipMemcpyHostToDevice) );
            cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_orienEC, orienEC, npeaks*sizeof(uint16_t), hipMemcpyHostToDevice) );
        //}

        //if (gpu_segmentsPerBlockEC == NULL || gpu_offsetPerBlockEC == NULL) {
            //uint32_t* segmentsPerBlock = (uint32_t*) malloc(nvoxels*sizeof(uint32_t));
            //uint32_t* offsetPerBlock   = (uint32_t*) malloc(nvoxels*sizeof(uint32_t));
            preprocessDataForGPU(voxelEC, npeaks, segmentsPerBlock, offsetPerBlock, nvoxels);

            cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_segmentsPerBlockEC, nvoxels*sizeof(uint32_t)) );
            cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_offsetPerBlockEC,   nvoxels*sizeof(uint32_t)) );

            cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_segmentsPerBlockEC, segmentsPerBlock, nvoxels*sizeof(uint32_t), hipMemcpyHostToDevice) );
            cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_offsetPerBlockEC,   offsetPerBlock,   nvoxels*sizeof(uint32_t), hipMemcpyHostToDevice) );

            free(segmentsPerBlock);
            free(offsetPerBlock);
        //}
    }
}

CudaLinearOperator::~CudaLinearOperator(){
    /*cudaStatus = cudaStatus && cudaCheck( hipFree(voxelIC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(fiberIC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(orienIC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(lengthIC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(lutIC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(segmentsPerBlockIC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(offsetPerBlockIC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(voxelEC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(orienEC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(lutEC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(segmentsPerBlockEC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(offsetPerBlockEC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(lutISO) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(TvoxelIC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(TfiberIC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(TorienIC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(TlengthIC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(TfibersPerBlockIC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(ToffsetPerBlockIC) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(x) );
    cudaStatus = cudaStatus && cudaCheck( hipFree(y) );

    cudaStatus = cudaStatus && cudaCheck( hipDeviceReset() );//*/
}

void CudaLinearOperator::destroy(){
    printf("\t* destroying and reseting GPU ... ");
    bool status = true;
    status = status && cudaCheck( hipFree(gpu_voxelIC) );
    status = status && cudaCheck( hipFree(gpu_fiberIC) );
    status = status && cudaCheck( hipFree(gpu_orienIC) );
    status = status && cudaCheck( hipFree(gpu_lengthIC) );
    status = status && cudaCheck( hipFree(gpu_lutIC) );
    status = status && cudaCheck( hipFree(gpu_segmentsPerBlockIC) );
    status = status && cudaCheck( hipFree(gpu_offsetPerBlockIC) );
    status = status && cudaCheck( hipFree(gpu_voxelEC) );
    status = status && cudaCheck( hipFree(gpu_orienEC) );
    status = status && cudaCheck( hipFree(gpu_lutEC) );
    status = status && cudaCheck( hipFree(gpu_segmentsPerBlockEC) );
    status = status && cudaCheck( hipFree(gpu_offsetPerBlockEC) );
    status = status && cudaCheck( hipFree(gpu_lutISO) );
    status = status && cudaCheck( hipFree(gpu_TvoxelIC) );
    status = status && cudaCheck( hipFree(gpu_TfiberIC) );
    status = status && cudaCheck( hipFree(gpu_TorienIC) );
    status = status && cudaCheck( hipFree(gpu_TlengthIC) );
    status = status && cudaCheck( hipFree(gpu_TfibersPerBlockIC) );
    status = status && cudaCheck( hipFree(gpu_ToffsetPerBlockIC) );
    status = status && cudaCheck( hipFree(gpu_x) );
    status = status && cudaCheck( hipFree(gpu_y) );

    /*printf("\t* reseting GPU ... ");
    bool status = true;//*/
    status = status && cudaCheck( hipDeviceReset() );
    if (status) printf("[ OK ]\n");
    else        printf("[ CUDA ERROR ]\n");//*/
}

void CudaLinearOperator::setTransposeData(
    uint32_t*  voxelIDs,
    uint32_t*  fiberIDs,
    uint16_t*  orienIDs,
    float32_t* lengths)
{
    //if (gpu_TvoxelIC == NULL || gpu_TfiberIC == NULL || gpu_TorienIC == NULL || gpu_TlengthIC == NULL) {
        cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_TvoxelIC,  nsegments*sizeof(uint32_t))  );
        cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_TfiberIC,  nsegments*sizeof(uint32_t))  );
        cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_TorienIC,  nsegments*sizeof(uint16_t))  );
        cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_TlengthIC, nsegments*sizeof(float32_t)) );

        cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_TvoxelIC,  voxelIDs, nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_TfiberIC,  fiberIDs, nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_TorienIC,  orienIDs, nsegments*sizeof(uint16_t),  hipMemcpyHostToDevice) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_TlengthIC, lengths,  nsegments*sizeof(float32_t), hipMemcpyHostToDevice) );
    //}
    
    //if (gpu_TfibersPerBlockIC == NULL || gpu_ToffsetPerBlockIC == NULL) {
        uint32_t*  fibersPerBlock = (uint32_t*) malloc(nfibers*sizeof(uint32_t));
        uint32_t*  offsetPerBlock = (uint32_t*) malloc(nfibers*sizeof(uint32_t));
    
        preprocessDataForGPU(fiberIDs, nsegments, fibersPerBlock, offsetPerBlock, nfibers);
    
        cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_TfibersPerBlockIC, nfibers*sizeof(uint32_t)) );
        cudaStatus = cudaStatus && cudaCheck( hipMalloc((void**)&gpu_ToffsetPerBlockIC, nfibers*sizeof(uint32_t)) );
    
        cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_TfibersPerBlockIC, fibersPerBlock, nfibers*sizeof(uint32_t),  hipMemcpyHostToDevice) );
        cudaStatus = cudaStatus && cudaCheck( hipMemcpy(gpu_ToffsetPerBlockIC, offsetPerBlock, nfibers*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    
        free(fibersPerBlock);
        free(offsetPerBlock);
    //}
}

void cudaCheckKernel(){
    hipError_t cudaStatus;
    
    cudaStatus = hipGetLastError();
	if(cudaStatus != hipSuccess)
        fprintf(stderr, "\t* kernel launch... [ ERROR ]: %s\n\n", hipGetErrorString(cudaStatus));
    else
        printf("\t* kernel launch... [ OK ]\n");

    cudaStatus = hipDeviceSynchronize();
	if(cudaStatus != hipSuccess)
        fprintf(stderr, "\t* hipDeviceSynchronize() after launching kernel... [ ERROR ]: %d\n", cudaStatus);
    else
        printf("\t* hipDeviceSynchronize() after launching kernel... [ OK ]\n");
}

void CudaLinearOperator::dot(float64_t* v_in, float64_t* v_out){
    //hipError_t cudaStatus;
    
    // Copy vector x to the GPU
    cudaStatus = hipMemcpy(gpu_x, v_in, ncols*sizeof(double), hipMemcpyHostToDevice);
    /*if (cudaStatus != hipSuccess) printf("\t* tranfering x to GPU ... [ ERROR ]: %s\n", hipGetErrorString(cudaStatus));
    else                           printf("\t* tranfering x to GPU ... [   OK  ]\n");//*/

    // Multiply IC part in the GPU
    multiply_Ax_ICpart<<<nvoxels, 1024>>>(gpu_voxelIC, gpu_fiberIC, gpu_orienIC, gpu_lengthIC, gpu_segmentsPerBlockIC, gpu_offsetPerBlockIC, gpu_lutIC, gpu_x, gpu_y);

    //cudaCheckKernel();

    // Multiply EC part in the GPU
    multiply_Ax_ECpart<<<nvoxels, 512>>>(gpu_voxelEC, gpu_orienEC, gpu_segmentsPerBlockEC, gpu_offsetPerBlockEC, gpu_lutEC, gpu_x, gpu_y);

    //cudaCheckKernel();

    // Multiply ISO part in the GPU
    multiply_Ax_ISOpart<<<nvoxels, 512>>>(gpu_lutISO, gpu_x, gpu_y);

    //cudaCheckKernel();

    // Copy back result to CPU
    cudaStatus = hipMemcpy(v_out, gpu_y, nrows*sizeof(double), hipMemcpyDeviceToHost);
    /*if (cudaStatus != hipSuccess) printf("\t* tranfering y to CPU ... [ ERROR ]: %s\n", hipGetErrorString(cudaStatus));
    else                           printf("\t* tranfering y to CPU ... [   OK  ]\n");//*/
}

void CudaLinearOperator::Tdot(float64_t* v_in, float64_t* v_out){
        
    //hipError_t cudaStatus;
    // Copy vector y to the GPU
    //cudaCheck( hipMemset(gpu_x, 0, NUM_COLS*sizeof(float64_t)) );
    //cudaCheck( hipMemcpy(gpu_x, x, NUM_COLS*sizeof(double), hipMemcpyHostToDevice) );
    cudaStatus = hipMemcpy(gpu_y, v_in, nrows*sizeof(double), hipMemcpyHostToDevice);
    /*if (cudaStatus != hipSuccess) printf("\t* tranfering y to GPU ... [ ERROR ]: %s\n", hipGetErrorString(cudaStatus));
    else                           printf("\t* tranfering y to GPU ... [   OK  ]\n");//*/

    // Multiply IC part in the GPU
    multiply_Aty_ICpart<<<nfibers, 512>>>(gpu_TvoxelIC, gpu_TfiberIC, gpu_TorienIC, gpu_TlengthIC, gpu_TfibersPerBlockIC, gpu_ToffsetPerBlockIC, gpu_lutIC, gpu_x, gpu_y);

    //cudaCheckKernel();

    // Multiply EC part in the GPU
    multiply_Aty_ECpart<<<nvoxels, 512>>>(gpu_voxelEC, gpu_orienEC, gpu_segmentsPerBlockEC, gpu_offsetPerBlockEC, gpu_lutEC, gpu_x, gpu_y);

    //cudaCheckKernel();

    // Multiply ISO part in the GPU
    multiply_Aty_ISOpart<<<nvoxels, 512>>>(gpu_lutISO, gpu_x, gpu_y);

    //cudaCheckKernel();

    // Copy back result to CPU
    cudaStatus = hipMemcpy(v_out, gpu_x, ncols*sizeof(double), hipMemcpyDeviceToHost);
    /*if (cudaStatus != hipSuccess) printf("\t* tranfering x to CPU ... [ ERROR ]: %s\n", hipGetErrorString(cudaStatus));
    else                           printf("\t* tranfering x to CPU ... [   OK  ]\n");//*/
        
    /*printf("\n\n VECTOR X EC PART:\n");
    for(int i = NUM_FIBERS*NUM_RESFUNCIC; i < NUM_FIBERS*NUM_RESFUNCIC+20; i++)
        printf("%lf ", x[i]);
    printf("\n\n");//*/
}

bool cudaCheck(hipError_t cudaStatus){
    return cudaStatus == hipSuccess;
}

void preprocessDataForGPU(uint32_t* data, int NUM_COMPARTMENTS, uint32_t* compartmentsPerBlock, uint32_t* offsetPerBlock, int NUM_BLOCKS){

    // fill arrays with zeros
    memset(compartmentsPerBlock, 0, NUM_BLOCKS * sizeof(uint32_t));
    memset(offsetPerBlock,       0, NUM_BLOCKS * sizeof(uint32_t));

    // count compartments per block
    for(int i = 0; i < NUM_COMPARTMENTS; i++)
        compartmentsPerBlock[data[i]]++;

    // calculate offset per block
    offsetPerBlock[0] = 0;
    for(int i = 1; i < NUM_BLOCKS; i++)
        offsetPerBlock[i] = offsetPerBlock[i-1] + compartmentsPerBlock[i-1];
}


__global__ void multiply_Ax_ICpart(
    uint32_t*  voxelIDs,
    uint32_t*  fiberIDs,
    uint16_t*  orienIDs,
    float32_t* lengths,
    uint32_t*  segmentsPerBlock,
    uint32_t*  offsetPerBlock,
    float32_t* lut,
    float64_t* x,
    float64_t* y){

    __shared__ float64_t shmem[1024];

    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;
    uint32_t gid = threadIdx.x / 512;
    uint32_t sid = threadIdx.x - 512*gid;

    shmem[tid] = 0.0;

    if(sid >= NUM_SAMPLES) return;

    uint32_t offset = offsetPerBlock[bid] + (segmentsPerBlock[bid]/2)*gid;
    uint32_t nsegments = segmentsPerBlock[bid]/2 + (segmentsPerBlock[bid]%2)*gid;

    //segment_t* segment = segments + offset;
    uint32_t*  voxel  = voxelIDs + offset;
    uint32_t*  fiber  = fiberIDs + offset;
    uint16_t*  orien  = orienIDs + offset;
    float32_t* length = lengths  + offset;

    float64_t sum = 0.0;

    for(int i = 0; i < nsegments; i++){
        int offset_lut = (*orien)*NUM_SAMPLES + sid;

        float64_t aux = 0.0;
        for(int j = 0; j < NUM_DIAMETERS; j++){
            aux += (double)(lut[offset_lut + j*NUM_ORIENTATIONS*NUM_SAMPLES])*x[(*fiber) + j*NUM_FIBERS];
            //aux += tex1Dfetch(tex_lutIC, offset_lut + j*num_orientations*num_samples) * x[(*fiber) + j*num_fibers];
        }

        sum += aux * (*length);

        fiber++;
        orien++;
        length++;
    }

    shmem[tid] = sum;
    __syncthreads();

    if(tid < NUM_SAMPLES)
        y[(*voxel)*NUM_SAMPLES + sid] = sum + shmem[tid+512];
}

__global__ void multiply_Ax_ECpart(
    uint32_t*  voxelIDs,
    uint16_t*  orienIDs,
    uint32_t*  segmentsPerBlock,
    uint32_t*  offsetPerBlock,
    float32_t* lut,
    float64_t* x,
    float64_t* y)
{
    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;

    if(tid >= NUM_SAMPLES) return;

    uint32_t offset  = offsetPerBlock[bid];
    uint32_t nsegments = segmentsPerBlock[bid];

    //compartmentEC_t* excomp = excomps + offset;
    uint32_t* voxel = voxelIDs + offset;
    uint16_t* orien = orienIDs + offset;

    uint32_t target = NUM_FIBERS*NUM_DIAMETERS + offset;

    float64_t sum = 0.0;
    for(int i = 0; i < nsegments; i++){
        uint32_t offset_lut = (*orien)*NUM_SAMPLES + tid;

        for(int j = 0; j < NUM_ZEPPELINS; j++)
            sum += (double)(lut[offset_lut + j*NUM_ORIENTATIONS*NUM_SAMPLES])*x[target + j*NUM_PEAKS + i];
            //sum += tex1Dfetch(tex_lutEC, offset_lut + j*num_orientations*num_samples) * x[target + j*num_excomps + i];

        orien++;
    }

    y[(*voxel)*NUM_SAMPLES + tid] += sum;
}

__global__ void multiply_Ax_ISOpart(
    float32_t* lut,
    float64_t* x,
    float64_t* y)
{
    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;

    if(tid >= NUM_SAMPLES) return;

    uint32_t target = NUM_FIBERS*NUM_DIAMETERS + NUM_PEAKS*NUM_ZEPPELINS + bid;

    float64_t sum = 0.0;
    for(int j = 0; j < NUM_BALLS; j++)
        sum += (double)(lut[j*NUM_SAMPLES + tid])*x[target + j*NUM_VOXELS];
        //sum += (double)(tex1Dfetch(tex_lutISO, j*num_samples + tid))*x[target + j*num_voxels];
        

    y[bid*NUM_SAMPLES + tid] += sum;
}

__global__ void multiply_Aty_ICpart(
    uint32_t*  voxelICt,
    uint32_t*  fiberICt,
    uint16_t*  orienICt,
    float32_t* lengthICt,
    uint32_t*  compartmentsPerBlock,
    uint32_t*  offsetPerBlock,
    float32_t* lut,
    float64_t* x,
    float64_t* y)
{
    __shared__ float64_t shmem[512];

    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;

    shmem[tid] = 0.0;

    if(tid >= NUM_SAMPLES) return;

    /*if(bid == 0 && tid == 0){
    for(int i = 0; i < 10; i++){
    printf("%d %d %d %f\n", voxelICt[i], fiberICt[i], orientICt[i], lengthICt[i]);
    }
    }
    else if(bid != 0) return;
    //__syncthreads();//*/

    uint32_t offset = offsetPerBlock[bid];
    uint32_t nsegments = offset + compartmentsPerBlock[bid];

    //segment_t* segment = segments + offset;
    uint32_t*  voxel  = voxelICt  + offset;
    uint32_t*  fiber  = fiberICt  + offset;
    uint16_t*  orien  = orienICt  + offset;
    float32_t* length = lengthICt + offset;
    //uint fiber = segment->fiber;

    for(int j = 0; j < NUM_DIAMETERS; j++){
        int offset_lut = j*NUM_ORIENTATIONS*NUM_SAMPLES + tid;

        float64_t sum = 0.0;
        //segment = segments + offset;
        voxel  = voxelICt  + offset;
        orien  = orienICt  + offset;
        length = lengthICt + offset;
        for(int i = offset; i < nsegments; i++){
            sum += ((float64_t)(*length)) *( (float64_t) lut[offset_lut + (*orien)*NUM_SAMPLES] )* y[(*voxel)*NUM_SAMPLES + tid];
            //sum += ((float64_t)(*length)) *( (float64_t) tex1Dfetch(tex_lutIC, offset_lut + (*orient)*num_samples) )* y[(*voxel)*num_samples + tid];
            //segment++;
            voxel++;
            //fiber++;
            orien++;
            length++;
        }

        shmem[tid] = sum;
        __syncthreads();

        if(tid < 256) shmem[tid] += shmem[tid + 256]; __syncthreads();
        if(tid < 128) shmem[tid] += shmem[tid + 128]; __syncthreads();
        if(tid <  64) shmem[tid] += shmem[tid +  64]; __syncthreads();
        if(tid <  32) shmem[tid] += shmem[tid +  32]; __syncthreads();
        if(tid <  16) shmem[tid] += shmem[tid +  16]; __syncthreads();
        if(tid <   8) shmem[tid] += shmem[tid +   8]; __syncthreads();
        if(tid <   4) shmem[tid] += shmem[tid +   4]; __syncthreads();
        //if(tid <   2) shmem[tid] += shmem[tid +   2]; __syncthreads();

        if(tid == 0) x[j*NUM_FIBERS + (*fiber)] = shmem[0] + shmem[1] + shmem[2] + shmem[3];

        __syncthreads();
    }
}

__global__ void multiply_Aty_ECpart(
    uint32_t*  voxelEC,
    uint16_t*  orienEC,
    uint32_t*  segmentsPerBlock,
    uint32_t*  offsetPerBlock,
    float32_t* lut,
    float64_t* x,
    float64_t* y)
{
    __shared__ float64_t shmem[512];

    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;

    shmem[tid] = 0.0;

    if(tid >= NUM_SAMPLES) return;

    uint32_t offset  = offsetPerBlock[bid];
    uint32_t ncompartments = segmentsPerBlock[bid] + offset;

    //compartmentEC_t* peak = peaks + offset;
    uint32_t* voxel = voxelEC + offset;
    uint16_t* orien = orienEC + offset;

    for(int j = 0; j < NUM_ZEPPELINS; j++){        
        uint32_t offset_lut = j*NUM_ORIENTATIONS*NUM_SAMPLES + tid;

        //peak = peaks + offset;
        voxel = voxelEC + offset;
        orien = orienEC + offset;
        for(int i = offset; i < ncompartments; i++){
            //shmem[tid] =( (float64_t)tex1Dfetch(tex_lutEC, (*orient)*num_samples + offset_lut) )* y[(*voxel)*num_samples + tid];
            shmem[tid] =( (float64_t)(lut[(*orien)*NUM_SAMPLES + offset_lut] ))* y[(*voxel)*NUM_SAMPLES + tid];
            __syncthreads();

            //if(bid == 0){
            //printf("%lf\n", lut[(peak->orientation)*num_samples + lut_offset] * y[(peak->voxel)*num_samples + tid]);

            if(tid < 256) shmem[tid] += shmem[tid + 256]; __syncthreads();
            if(tid < 128) shmem[tid] += shmem[tid + 128]; __syncthreads();
            if(tid <  64) shmem[tid] += shmem[tid +  64]; __syncthreads();
            if(tid <  32) shmem[tid] += shmem[tid +  32]; __syncthreads();
            if(tid <  16) shmem[tid] += shmem[tid +  16]; __syncthreads();
            if(tid <   8) shmem[tid] += shmem[tid +   8]; __syncthreads();
            if(tid <   4) shmem[tid] += shmem[tid +   4]; __syncthreads();
            if(tid <   2) shmem[tid] += shmem[tid +   2]; __syncthreads();

            if(tid == 0) x[NUM_FIBERS*NUM_DIAMETERS + j*NUM_PEAKS + i] = shmem[0] + shmem[1];
            //}

            //peak++;
            voxel++;
            orien++;
            __syncthreads();
        }
    }
} //*/

__global__ void multiply_Aty_ISOpart(float* lut, double* x, double* y){
    __shared__ double shmem[512];

    uint bid = blockIdx.x;
    uint tid = threadIdx.x;
    uint offset = NUM_FIBERS*NUM_DIAMETERS + NUM_PEAKS*NUM_ZEPPELINS + bid;

    shmem[tid] = 0.0;

    if(tid >= NUM_SAMPLES) return;

    for(int j = 0; j < NUM_BALLS; j++){
        shmem[tid] =( (float64_t) lut[j*NUM_SAMPLES + tid] )* y[bid*NUM_SAMPLES + tid];
        //shmem[tid] =( (float64_t) tex1Dfetch(tex_lutISO, j*num_samples + tid) )* y[bid*num_samples + tid];
        __syncthreads();

        if(tid < 256) shmem[tid] += shmem[tid + 256]; __syncthreads();
        if(tid < 128) shmem[tid] += shmem[tid + 128]; __syncthreads();
        if(tid <  64) shmem[tid] += shmem[tid +  64]; __syncthreads();
        if(tid <  32) shmem[tid] += shmem[tid +  32]; __syncthreads();
        if(tid <  16) shmem[tid] += shmem[tid +  16]; __syncthreads();
        if(tid <   8) shmem[tid] += shmem[tid +   8]; __syncthreads();
        if(tid <   4) shmem[tid] += shmem[tid +   4]; __syncthreads(); 

        if(tid == 0)
            x[offset + j*NUM_VOXELS] = shmem[0] + shmem[1] + shmem[2] + shmem[3];
    }
}//*/


#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <algorithm>

using namespace std;

typedef unsigned int uint32_t;
typedef unsigned short int uint16_t;
typedef float float32_t;
typedef double float64_t;

bool cudaCheck(hipError_t cudaStatus);
void preprocessDataForGPU(uint32_t* data, int NUM_COMPARTMENTS, uint32_t* compartmentsPerBlock, uint32_t* offsetPerBlock, int NUM_BLOCKS);

// constant values in GPU
__constant__ int NUM_VOXELS;
__constant__ int NUM_FIBERS;
__constant__ int NUM_PEAKS;
__constant__ int NUM_ORIENTATIONS;
__constant__ int NUM_SAMPLES;
__constant__ int NUM_DIAMETERS;
__constant__ int NUM_ZEPPELINS;
__constant__ int NUM_BALLS;
__constant__ int NUM_ROWS;        
__constant__ int NUM_COLS;      
__constant__ int SIZE_LUTIC;      
__constant__ int SIZE_LUTEC;     
__constant__ int SIZE_LUTISO;

class CudaLinearOperator {

    // pointers to IC data in GPU memory
    uint32_t*  voxelIC;
    uint32_t*  fiberIC;
    uint16_t*  orienIC;
    float32_t* lengthIC;

    // pointers to IC data (transpose) in GPU memory
    uint32_t*  voxelICt;
    uint32_t*  fiberICt;
    uint16_t*  orienICt;
    float32_t* lengthICt;
    uint32_t* fibersPerBlockICt;
    uint32_t* offsetPerBlockICt;

    // auxiliar arrays for GPU
    uint32_t* segmentsPerBlockIC;
    uint32_t* offsetPerBlockIC;
    uint32_t* segmentsPerBlockEC;
    uint32_t* offsetPerBlockEC;

    // pointers to EC data in GPU memory
    uint32_t*  voxelEC;
    uint16_t*  orienEC;

    // pointers to LUTs in GPU memory
    float32_t* lutIC;
    float32_t* lutEC;
    float32_t* lutISO;

    // pointers to vector x and y
    float64_t* x;
    float64_t* y;

    // dimensions of the operator
    int nrows;
    int ncols;
    int nvoxels;
    int nfibers;

    public:
        CudaLinearOperator(
            uint32_t* voxelIC,
            uint32_t* fiberIC,
            uint16_t* orienIC,
            float*    lengthIC,
            float*    lutIC,
        
            uint32_t* voxelEC,
            uint16_t* orienEC,
            float*    lutEC,
        
            float*    lutISO,
        
            int nsegments,
            int nvoxels,      
            int nfibers,      
            int npeaks,       
            int norientations,
            int nsamples,     
            int ndiameters,   
            int nzeppelins,   
            int nballs)
        {
            this->nvoxels = nvoxels;
            this->nfibers = nfibers;
            this->nrows = nvoxels * nsamples;
            this->ncols = nfibers*ndiameters + npeaks*nzeppelins + nvoxels*nballs;
            int size_lutic  = ndiameters*norientations*nsamples;
            int size_lutec  = nzeppelins*norientations*nsamples;
            int size_lutiso = nballs*nsamples;
            bool status;
        
            uint32_t* segmentsPerBlock = (uint32_t*) malloc(nvoxels*sizeof(uint32_t));
            uint32_t* offsetPerBlock   = (uint32_t*) malloc(nvoxels*sizeof(uint32_t));
        
            // copy constant values to GPU
            printf("\t* constant global values ... ");
            status = true;
            status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_VOXELS),       &nvoxels,       sizeof(int)) );
            status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_FIBERS),       &nfibers,       sizeof(int)) );
            status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_PEAKS),        &npeaks,        sizeof(int)) );
            status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_ORIENTATIONS), &norientations, sizeof(int)) );
            status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_SAMPLES),      &nsamples,      sizeof(int)) );
            status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_DIAMETERS),    &ndiameters,    sizeof(int)) );
            status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_ZEPPELINS),    &nzeppelins,    sizeof(int)) );
            status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_BALLS),        &nballs,        sizeof(int)) );
            status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_ROWS),         &nrows,         sizeof(int)) );
            status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_COLS),         &ncols,         sizeof(int)) );
            status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTIC),       &size_lutic,    sizeof(int)) );
            status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTEC),       &size_lutec,    sizeof(int)) );
            status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTISO),      &size_lutiso,   sizeof(int)) );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        
        
            // alloc memory in GPU for vectors x and y
            printf("\t* memory for vectors x and y ... ");
            status = true;
            status = status && cudaCheck( hipMalloc((void**)&(this->x), ncols*sizeof(float64_t)) );
            status = status && cudaCheck( hipMalloc((void**)&(this->y), nrows*sizeof(float64_t)) );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        
            // alloc GPU memory for segments
            printf("\t* memory for LUT (IC part) ... ");
            status = true;
            status = status && cudaCheck( hipMalloc((void**)&(this->lutIC), size_lutic*sizeof(float32_t)) );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        
            printf("\t* copying LUT in GPU (IC part) ... ");
            status = true;
            status = status && cudaCheck( hipMemcpy(this->lutIC, lutIC, size_lutic*sizeof(float32_t), hipMemcpyHostToDevice) );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        
            printf("\t* allocating memory for LUT in GPU (EC part) ... ");
            status = cudaCheck( hipMalloc((void**)&(this->lutEC), size_lutec*sizeof(float32_t)) );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        
            printf("\t* copying LUT in GPU (EC part) ... ");
            status = cudaCheck( hipMemcpy(this->lutEC, lutEC, size_lutec*sizeof(float32_t), hipMemcpyHostToDevice) );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        
            printf("\t* allocating memory for LUT in GPU (ISO part) ... ");
            status = cudaCheck( hipMalloc((void**)&(this->lutISO), size_lutiso*sizeof(float32_t)) );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        
            printf("\t* copying LUT in GPU (ISO part) ... ");
            status = cudaCheck( hipMemcpy(this->lutISO, lutISO, size_lutiso*sizeof(float32_t), hipMemcpyHostToDevice) );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        
            printf("\t* preprocessing data for GPU ... ");
            preprocessDataForGPU(voxelIC, nsegments, segmentsPerBlock, offsetPerBlock, nvoxels);
            printf("\n");
        
            printf("\t* fiber segments memory allocation ... ");
            status = true;
            status = status && cudaCheck( hipMalloc((void**)&(this->voxelIC),  nsegments*sizeof(uint32_t))  );
            status = status && cudaCheck( hipMalloc((void**)&(this->fiberIC),  nsegments*sizeof(uint32_t))  );
            status = status && cudaCheck( hipMalloc((void**)&(this->orienIC),  nsegments*sizeof(uint16_t))  );
            status = status && cudaCheck( hipMalloc((void**)&(this->lengthIC), nsegments*sizeof(float32_t)) );
            status = status && cudaCheck( hipMalloc((void**)&(this->segmentsPerBlockIC), nvoxels*sizeof(uint32_t)) );
            status = status && cudaCheck( hipMalloc((void**)&(this->offsetPerBlockIC),   nvoxels*sizeof(uint32_t)) );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        
            printf("\t* transfering fiber segments ... ");
            status = true;
            status = status && cudaCheck( hipMemcpy(this->voxelIC,  voxelIC,  nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice) );
            status = status && cudaCheck( hipMemcpy(this->fiberIC,  fiberIC,  nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice) );
            status = status && cudaCheck( hipMemcpy(this->orienIC,  orienIC,  nsegments*sizeof(uint16_t),  hipMemcpyHostToDevice) );
            status = status && cudaCheck( hipMemcpy(this->lengthIC, lengthIC, nsegments*sizeof(float32_t), hipMemcpyHostToDevice) );
            status = status && cudaCheck( hipMemcpy(this->segmentsPerBlockIC, segmentsPerBlock, nvoxels*sizeof(uint32_t),  hipMemcpyHostToDevice) );
            status = status && cudaCheck( hipMemcpy(this->offsetPerBlockIC,   offsetPerBlock,   nvoxels*sizeof(uint32_t),  hipMemcpyHostToDevice) );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        
            // ---------------------------------------- EC DATA ---------------------------------------- //
            printf("\t* allocating memory for operator A in GPU (EC part) ... ");
            status = true;
            status = status && cudaCheck( hipMalloc((void**)&(this->voxelEC),  npeaks*sizeof(uint32_t)) );
            status = status && cudaCheck( hipMalloc((void**)&(this->orienEC),  npeaks*sizeof(uint16_t)) );
            status = status && cudaCheck( hipMalloc((void**)&(this->segmentsPerBlockEC), nvoxels*sizeof(uint32_t))  );
            status = status && cudaCheck( hipMalloc((void**)&(this->offsetPerBlockEC),   nvoxels*sizeof(uint32_t))  );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        
            printf("\t* preprocessing EC data for GPU ... ");
            preprocessDataForGPU(voxelEC, npeaks, segmentsPerBlock, offsetPerBlock, nvoxels);
            printf("\n");
        
            printf("\t* copying operator A to GPU (EC part) ... ");
            status = true;
            status = status && cudaCheck( hipMemcpy(this->voxelEC,            voxelEC,              npeaks*sizeof(uint32_t),  hipMemcpyHostToDevice) );
            status = status && cudaCheck( hipMemcpy(this->orienEC,            orienEC,              npeaks*sizeof(uint16_t),  hipMemcpyHostToDevice) );
            status = status && cudaCheck( hipMemcpy(this->segmentsPerBlockEC, segmentsPerBlock,     nvoxels*sizeof(uint32_t), hipMemcpyHostToDevice) );
            status = status && cudaCheck( hipMemcpy(this->offsetPerBlockEC,   offsetPerBlock,       nvoxels*sizeof(uint32_t), hipMemcpyHostToDevice) );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        
            free(segmentsPerBlock);
            free(offsetPerBlock);
        }
        
        ~CudaLinearOperator(){
            hipFree(voxelIC);
            hipFree(fiberIC);
            hipFree(orienIC);
            hipFree(lengthIC);
            hipFree(lutIC);
            hipFree(segmentsPerBlockIC);
            hipFree(offsetPerBlockIC);
            
            hipFree(voxelEC);
            hipFree(orienEC);
            hipFree(lutEC);
            hipFree(segmentsPerBlockEC);
            hipFree(offsetPerBlockEC);
        
            hipFree(lutISO);
        
            hipFree(voxelICt);
            hipFree(fiberICt);
            hipFree(orienICt);
            hipFree(lengthICt);
            hipFree(fibersPerBlockICt);
            hipFree(offsetPerBlockICt);
        
            hipFree(x);
            hipFree(y);
        
            printf("\t* reseting GPU ... ");
            bool status = true;
            status = status && cudaCheck( hipDeviceReset() );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        }

        void setTransposeData(
            uint32_t*  voxelIDs,
            uint32_t*  fiberIDs,
            uint16_t*  orienIDs,
            float32_t* lengths,
            int nsegments)
        {
            bool status;
            uint32_t*  fibersPerBlock = (uint32_t*) malloc(nfibers*sizeof(uint32_t));
            uint32_t*  offsetPerBlock = (uint32_t*) malloc(nfibers*sizeof(uint32_t));
        
            preprocessDataForGPU(fiberIDs, nsegments, fibersPerBlock, offsetPerBlock, nfibers);
        
            printf("\t* extra memory for operator A' ... ");
            status = true;
            status = status && cudaCheck( hipMalloc((void**)&(voxelICt),  nsegments*sizeof(uint32_t))  );
            status = status && cudaCheck( hipMalloc((void**)&(fiberICt),  nsegments*sizeof(uint32_t))  );
            status = status && cudaCheck( hipMalloc((void**)&(orienICt),  nsegments*sizeof(uint16_t))  );
            status = status && cudaCheck( hipMalloc((void**)&(lengthICt), nsegments*sizeof(float32_t)) );
            status = status && cudaCheck( hipMalloc((void**)&(fibersPerBlockICt), nfibers*sizeof(uint32_t)) );
            status = status && cudaCheck( hipMalloc((void**)&(offsetPerBlockICt), nfibers*sizeof(uint32_t)) );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        
            printf("\t* transfering memory for operator A' ... ");
            status = true;
            status = status && cudaCheck( hipMemcpy(voxelICt,  voxelIDs, nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice) );
            status = status && cudaCheck( hipMemcpy(fiberICt,  fiberIDs, nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice) );
            status = status && cudaCheck( hipMemcpy(orienICt,  orienIDs, nsegments*sizeof(uint16_t),  hipMemcpyHostToDevice) );
            status = status && cudaCheck( hipMemcpy(lengthICt, lengths,  nsegments*sizeof(float32_t), hipMemcpyHostToDevice) );
            status = status && cudaCheck( hipMemcpy(fibersPerBlockICt, fibersPerBlock, nfibers*sizeof(uint32_t),  hipMemcpyHostToDevice) );
            status = status && cudaCheck( hipMemcpy(offsetPerBlockICt, offsetPerBlock, nfibers*sizeof(uint32_t),  hipMemcpyHostToDevice) );
            if (status) printf("[ OK ]\n");
            else        printf("[ ERROR ]\n");
        
            free(fibersPerBlock);
            free(offsetPerBlock);
        }

        void multiplyByX(float64_t* x, float64_t* y){
            // Copy vector x to the GPU
            hipMemcpy(this->x, x, ncols*sizeof(double), hipMemcpyHostToDevice);

            // Multiply IC part in the GPU
            multiply_Ax_ICpart<<<nvoxels, 1024>>>(voxelIC, fiberIC, orienIC, lengthIC, segmentsPerBlockIC, offsetPerBlockIC, lutIC, this->x, this->y);

            //cudaCheckKernel();

            // Multiply EC part in the GPU
            multiply_Ax_ECpart<<<nvoxels, 512>>>(voxelEC, orienEC, segmentsPerBlockEC, offsetPerBlockEC, lutEC, this->x, this->y);

            //cudaCheckKernel();

            // Multiply ISO part in the GPU
            multiply_Ax_ISOpart<<<nvoxels, 512>>>(lutISO, this->x, this->y);

            //cudaCheckKernel();

            // Copy back result to CPU
            hipMemcpy(y, this->y, nrows*sizeof(double), hipMemcpyDeviceToHost);
        }

        void multiplyByY(float64_t* v_in, float64_t* v_out){
        
            // Copy vector y to the GPU
            //cudaCheck( hipMemset(gpu_x, 0, NUM_COLS*sizeof(float64_t)) );
            //cudaCheck( hipMemcpy(gpu_x, x, NUM_COLS*sizeof(double), hipMemcpyHostToDevice) );
            cudaCheck( hipMemcpy(y, v_in, nrows*sizeof(double), hipMemcpyHostToDevice) );
        
            // Multiply IC part in the GPU
            multiply_Aty_ICpart<<<nfibers, 512>>>(voxelICt, fiberICt, orienICt, lengthICt, fibersPerBlockICt, offsetPerBlockICt, lutIC, x, y);
        
            //cudaCheckKernel();//*/
        
            // Multiply EC part in the GPU
            multiply_Aty_ECpart<<<nvoxels, 512>>>(voxelEC, orienEC, segmentsPerBlockEC, offsetPerBlockEC, lutEC, x, y);
        
            //cudaCheckKernel();
        
            // Multiply ISO part in the GPU
            multiply_Aty_ISOpart<<<nvoxels, 512>>>(lutISO, x, y);
        
            //cudaCheckKernel();//*/
        
            // Copy back result to CPU
            cudaCheck( hipMemcpy(v_out, x, ncols*sizeof(double), hipMemcpyDeviceToHost) );
                
            /*printf("\n\n VECTOR X EC PART:\n");
            for(int i = NUM_FIBERS*NUM_RESFUNCIC; i < NUM_FIBERS*NUM_RESFUNCIC+20; i++)
                printf("%lf ", x[i]);
            printf("\n\n");//*/
        }
};

bool cudaCheck(hipError_t cudaStatus){
    return cudaStatus == hipSuccess;
}

void preprocessDataForGPU(uint32_t* data, int NUM_COMPARTMENTS, uint32_t* compartmentsPerBlock, uint32_t* offsetPerBlock, int NUM_BLOCKS){

    // fill arrays with zeros
    memset(compartmentsPerBlock, 0, NUM_BLOCKS * sizeof(uint32_t));
    memset(offsetPerBlock,       0, NUM_BLOCKS * sizeof(uint32_t));

    // count compartments per block
    for(int i = 0; i < NUM_COMPARTMENTS; i++)
        compartmentsPerBlock[data[i]]++;

    // calculate offset per block
    offsetPerBlock[0] = 0;
    for(int i = 1; i < NUM_BLOCKS; i++)
        offsetPerBlock[i] = offsetPerBlock[i-1] + compartmentsPerBlock[i-1];
}

/*CudaLinearOperator::CudaLinearOperator(
    uint32_t* voxelIC,
    uint32_t* fiberIC,
    uint16_t* orienIC,
    float*    lengthIC,
    float*    lutIC,

    uint32_t* voxelEC,
    uint16_t* orienEC,
    float*    lutEC,

    float*    lutISO,

    int nsegments,
    int nvoxels,      
    int nfibers,      
    int npeaks,       
    int norientations,
    int nsamples,     
    int ndiameters,   
    int nzeppelins,   
    int nballs)
{
    this->nvoxels = nvoxels;
    this->nfibers = nfibers;
    this->nrows = nvoxels * nsamples;
    this->ncols = nfibers*ndiameters + npeaks*nzeppelins + nvoxels*nballs;
    int size_lutic  = ndiameters*norientations*nsamples;
    int size_lutec  = nzeppelins*norientations*nsamples;
    int size_lutiso = nballs*nsamples;
    bool status;

    uint32_t* segmentsPerBlock = (uint32_t*) malloc(nvoxels*sizeof(uint32_t));
    uint32_t* offsetPerBlock   = (uint32_t*) malloc(nvoxels*sizeof(uint32_t));

    // copy constant values to GPU
    printf("\t* constant global values ... ");
    status = true;
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_VOXELS),       &nvoxels,       sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_FIBERS),       &nfibers,       sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_PEAKS),        &npeaks,        sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_ORIENTATIONS), &norientations, sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_SAMPLES),      &nsamples,      sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_DIAMETERS),    &ndiameters,    sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_ZEPPELINS),    &nzeppelins,    sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_BALLS),        &nballs,        sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_ROWS),         &nrows,         sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(NUM_COLS),         &ncols,         sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTIC),       &size_lutic,    sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTEC),       &size_lutec,    sizeof(int)) );
    status = status && cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(SIZE_LUTISO),      &size_lutiso,   sizeof(int)) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");


    // alloc memory in GPU for vectors x and y
    printf("\t* memory for vectors x and y ... ");
    status = true;
    status = status && cudaCheck( hipMalloc((void**)&(this->x), ncols*sizeof(float64_t)) );
    status = status && cudaCheck( hipMalloc((void**)&(this->y), nrows*sizeof(float64_t)) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    // alloc GPU memory for segments
    printf("\t* memory for LUT (IC part) ... ");
    status = true;
    status = status && cudaCheck( hipMalloc((void**)&(this->lutIC), size_lutic*sizeof(float32_t)) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* copying LUT in GPU (IC part) ... ");
    status = true;
    status = status && cudaCheck( hipMemcpy(this->lutIC, lutIC, size_lutic*sizeof(float32_t), hipMemcpyHostToDevice) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* allocating memory for LUT in GPU (EC part) ... ");
    status = cudaCheck( hipMalloc((void**)&(this->lutEC), size_lutec*sizeof(float32_t)) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* copying LUT in GPU (EC part) ... ");
    status = cudaCheck( hipMemcpy(this->lutEC, lutEC, size_lutec*sizeof(float32_t), hipMemcpyHostToDevice) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* allocating memory for LUT in GPU (ISO part) ... ");
    status = cudaCheck( hipMalloc((void**)&(this->lutISO), size_lutiso*sizeof(float32_t)) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* copying LUT in GPU (ISO part) ... ");
    status = cudaCheck( hipMemcpy(this->lutISO, lutISO, size_lutiso*sizeof(float32_t), hipMemcpyHostToDevice) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* preprocessing data for GPU ... ");
    preprocessDataForGPU(voxelIC, nsegments, segmentsPerBlock, offsetPerBlock, nvoxels);
    printf("\n");

    printf("\t* fiber segments memory allocation ... ");
    status = true;
    status = status && cudaCheck( hipMalloc((void**)&(this->voxelIC),  nsegments*sizeof(uint32_t))  );
    status = status && cudaCheck( hipMalloc((void**)&(this->fiberIC),  nsegments*sizeof(uint32_t))  );
    status = status && cudaCheck( hipMalloc((void**)&(this->orienIC),  nsegments*sizeof(uint16_t))  );
    status = status && cudaCheck( hipMalloc((void**)&(this->lengthIC), nsegments*sizeof(float32_t)) );
    status = status && cudaCheck( hipMalloc((void**)&(this->segmentsPerBlockIC), nvoxels*sizeof(uint32_t)) );
    status = status && cudaCheck( hipMalloc((void**)&(this->offsetPerBlockIC),   nvoxels*sizeof(uint32_t)) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* transfering fiber segments ... ");
    status = true;
    status = status && cudaCheck( hipMemcpy(this->voxelIC,  voxelIC,  nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->fiberIC,  fiberIC,  nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->orienIC,  orienIC,  nsegments*sizeof(uint16_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->lengthIC, lengthIC, nsegments*sizeof(float32_t), hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->segmentsPerBlockIC, segmentsPerBlock, nvoxels*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->offsetPerBlockIC,   offsetPerBlock,   nvoxels*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    // ---------------------------------------- EC DATA ---------------------------------------- //
    printf("\t* allocating memory for operator A in GPU (EC part) ... ");
    status = true;
    status = status && cudaCheck( hipMalloc((void**)&(this->voxelEC),  npeaks*sizeof(uint32_t)) );
    status = status && cudaCheck( hipMalloc((void**)&(this->orienEC),  npeaks*sizeof(uint16_t)) );
    status = status && cudaCheck( hipMalloc((void**)&(this->segmentsPerBlockEC), nvoxels*sizeof(uint32_t))  );
    status = status && cudaCheck( hipMalloc((void**)&(this->offsetPerBlockEC),   nvoxels*sizeof(uint32_t))  );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* preprocessing EC data for GPU ... ");
    preprocessDataForGPU(voxelEC, npeaks, segmentsPerBlock, offsetPerBlock, nvoxels);
    printf("\n");

    printf("\t* copying operator A to GPU (EC part) ... ");
    status = true;
    status = status && cudaCheck( hipMemcpy(this->voxelEC,            voxelEC,              npeaks*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->orienEC,            orienEC,              npeaks*sizeof(uint16_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->segmentsPerBlockEC, segmentsPerBlock,     nvoxels*sizeof(uint32_t), hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(this->offsetPerBlockEC,   offsetPerBlock,       nvoxels*sizeof(uint32_t), hipMemcpyHostToDevice) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    free(segmentsPerBlock);
    free(offsetPerBlock);
}*/

/*CudaLinearOperator::~CudaLinearOperator(){
    hipFree(voxelIC);
    hipFree(fiberIC);
    hipFree(orienIC);
    hipFree(lengthIC);
    hipFree(lutIC);
    hipFree(segmentsPerBlockIC);
    hipFree(offsetPerBlockIC);
    
    hipFree(voxelEC);
    hipFree(orienEC);
    hipFree(lutEC);
    hipFree(segmentsPerBlockEC);
    hipFree(offsetPerBlockEC);

    hipFree(lutISO);

    hipFree(voxelICt);
    hipFree(fiberICt);
    hipFree(orienICt);
    hipFree(lengthICt);
    hipFree(fibersPerBlockICt);
    hipFree(offsetPerBlockICt);

    hipFree(x);
    hipFree(y);

    printf("\t* reseting GPU ... ");
    bool status = true;
    status = status && cudaCheck( hipDeviceReset() );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");
}*/

/*void CudaLinearOperator::setTransposeData(
    uint32_t*  voxelIDs,
    uint32_t*  fiberIDs,
    uint16_t*  orienIDs,
    float32_t* lengths,
    int nsegments)
{
    bool status;
    uint32_t*  fibersPerBlock = (uint32_t*) malloc(nfibers*sizeof(uint32_t));
    uint32_t*  offsetPerBlock = (uint32_t*) malloc(nfibers*sizeof(uint32_t));

    preprocessDataForGPU(fiberIDs, nsegments, fibersPerBlock, offsetPerBlock, nfibers);

    printf("\t* extra memory for operator A' ... ");
    status = true;
    status = status && cudaCheck( hipMalloc((void**)&(voxelICt),  nsegments*sizeof(uint32_t))  );
    status = status && cudaCheck( hipMalloc((void**)&(fiberICt),  nsegments*sizeof(uint32_t))  );
    status = status && cudaCheck( hipMalloc((void**)&(orienICt),  nsegments*sizeof(uint16_t))  );
    status = status && cudaCheck( hipMalloc((void**)&(lengthICt), nsegments*sizeof(float32_t)) );
    status = status && cudaCheck( hipMalloc((void**)&(fibersPerBlockICt), nfibers*sizeof(uint32_t)) );
    status = status && cudaCheck( hipMalloc((void**)&(offsetPerBlockICt), nfibers*sizeof(uint32_t)) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    printf("\t* transfering memory for operator A' ... ");
    status = true;
    status = status && cudaCheck( hipMemcpy(voxelICt,  voxelIDs, nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(fiberICt,  fiberIDs, nsegments*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(orienICt,  orienIDs, nsegments*sizeof(uint16_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(lengthICt, lengths,  nsegments*sizeof(float32_t), hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(fibersPerBlockICt, fibersPerBlock, nfibers*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    status = status && cudaCheck( hipMemcpy(offsetPerBlockICt, offsetPerBlock, nfibers*sizeof(uint32_t),  hipMemcpyHostToDevice) );
    if (status) printf("[ OK ]\n");
    else        printf("[ ERROR ]\n");

    free(fibersPerBlock);
    free(offsetPerBlock);
}*/

__global__ void multiply_Ax_ICpart(
    uint32_t*  voxelIDs,
    uint32_t*  fiberIDs,
    uint16_t*  orienIDs,
    float32_t* lengths,
    uint32_t*  segmentsPerBlock,
    uint32_t*  offsetPerBlock,
    float32_t* lut,
    float64_t* x,
    float64_t* y){

    __shared__ float64_t shmem[1024];

    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;
    uint32_t gid = threadIdx.x / 512;
    uint32_t sid = threadIdx.x - 512*gid;

    shmem[tid] = 0.0;

    if(sid >= NUM_SAMPLES) return;

    uint32_t offset = offsetPerBlock[bid] + (segmentsPerBlock[bid]/2)*gid;
    uint32_t nsegments = segmentsPerBlock[bid]/2 + (segmentsPerBlock[bid]%2)*gid;

    //segment_t* segment = segments + offset;
    uint32_t*  voxel  = voxelIDs + offset;
    uint32_t*  fiber  = fiberIDs + offset;
    uint16_t*  orien  = orienIDs + offset;
    float32_t* length = lengths  + offset;

    float64_t sum = 0.0;

    for(int i = 0; i < nsegments; i++){
        int offset_lut = (*orien)*NUM_SAMPLES + sid;

        float64_t aux = 0.0;
        for(int j = 0; j < NUM_DIAMETERS; j++){
            aux += (double)(lut[offset_lut + j*NUM_ORIENTATIONS*NUM_SAMPLES])*x[(*fiber) + j*NUM_FIBERS];
            //aux += tex1Dfetch(tex_lutIC, offset_lut + j*num_orientations*num_samples) * x[(*fiber) + j*num_fibers];
        }

        sum += aux * (*length);

        fiber++;
        orien++;
        length++;
    }

    shmem[tid] = sum;
    __syncthreads();

    if(tid < NUM_SAMPLES)
        y[(*voxel)*NUM_SAMPLES + sid] = sum + shmem[tid+512];
}

__global__ void multiply_Ax_ECpart(
    uint32_t*  voxelIDs,
    uint16_t*  orienIDs,
    uint32_t*  segmentsPerBlock,
    uint32_t*  offsetPerBlock,
    float32_t* lut,
    float64_t* x,
    float64_t* y)
{
    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;

    if(tid >= NUM_SAMPLES) return;

    uint32_t offset  = offsetPerBlock[bid];
    uint32_t nsegments = segmentsPerBlock[bid];

    //compartmentEC_t* excomp = excomps + offset;
    uint32_t* voxel = voxelIDs + offset;
    uint16_t* orien = orienIDs + offset;

    uint32_t target = NUM_FIBERS*NUM_DIAMETERS + offset;

    float64_t sum = 0.0;
    for(int i = 0; i < nsegments; i++){
        uint32_t offset_lut = (*orien)*NUM_SAMPLES + tid;

        for(int j = 0; j < NUM_ZEPPELINS; j++)
            sum += (double)(lut[offset_lut + j*NUM_ORIENTATIONS*NUM_SAMPLES])*x[target + j*NUM_PEAKS + i];
            //sum += tex1Dfetch(tex_lutEC, offset_lut + j*num_orientations*num_samples) * x[target + j*num_excomps + i];

        orien++;
    }

    y[(*voxel)*NUM_SAMPLES + tid] += sum;
}

__global__ void multiply_Ax_ISOpart(
    float32_t* lut,
    float64_t* x,
    float64_t* y)
{
    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;

    if(tid >= NUM_SAMPLES) return;

    uint32_t target = NUM_FIBERS*NUM_DIAMETERS + NUM_PEAKS*NUM_ZEPPELINS + bid;

    float64_t sum = 0.0;
    for(int j = 0; j < NUM_BALLS; j++)
        sum += (double)(lut[j*NUM_SAMPLES + tid])*x[target + j*NUM_VOXELS];
        //sum += (double)(tex1Dfetch(tex_lutISO, j*num_samples + tid))*x[target + j*num_voxels];
        

    y[bid*NUM_SAMPLES + tid] += sum;
}

__global__ void multiply_Aty_ICpart(
    uint32_t*  voxelICt,
    uint32_t*  fiberICt,
    uint16_t*  orienICt,
    float32_t* lengthICt,
    uint32_t*  compartmentsPerBlock,
    uint32_t*  offsetPerBlock,
    float32_t* lut,
    float64_t* x,
    float64_t* y)
{
    __shared__ float64_t shmem[512];

    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;

    shmem[tid] = 0.0;

    if(tid >= NUM_SAMPLES) return;

    /*if(bid == 0 && tid == 0){
    for(int i = 0; i < 10; i++){
    printf("%d %d %d %f\n", voxelICt[i], fiberICt[i], orientICt[i], lengthICt[i]);
    }
    }
    else if(bid != 0) return;
    //__syncthreads();//*/

    uint32_t offset = offsetPerBlock[bid];
    uint32_t nsegments = offset + compartmentsPerBlock[bid];

    //segment_t* segment = segments + offset;
    uint32_t*  voxel  = voxelICt  + offset;
    uint32_t*  fiber  = fiberICt  + offset;
    uint16_t*  orien  = orienICt  + offset;
    float32_t* length = lengthICt + offset;
    //uint fiber = segment->fiber;

    for(int j = 0; j < NUM_DIAMETERS; j++){
        int offset_lut = j*NUM_ORIENTATIONS*NUM_SAMPLES + tid;

        float64_t sum = 0.0;
        //segment = segments + offset;
        voxel  = voxelICt  + offset;
        orien  = orienICt  + offset;
        length = lengthICt + offset;
        for(int i = offset; i < nsegments; i++){
            sum += ((float64_t)(*length)) *( (float64_t) lut[offset_lut + (*orien)*NUM_SAMPLES] )* y[(*voxel)*NUM_SAMPLES + tid];
            //sum += ((float64_t)(*length)) *( (float64_t) tex1Dfetch(tex_lutIC, offset_lut + (*orient)*num_samples) )* y[(*voxel)*num_samples + tid];
            //segment++;
            voxel++;
            //fiber++;
            orien++;
            length++;
        }

        shmem[tid] = sum;
        __syncthreads();

        if(tid < 256) shmem[tid] += shmem[tid + 256]; __syncthreads();
        if(tid < 128) shmem[tid] += shmem[tid + 128]; __syncthreads();
        if(tid <  64) shmem[tid] += shmem[tid +  64]; __syncthreads();
        if(tid <  32) shmem[tid] += shmem[tid +  32]; __syncthreads();
        if(tid <  16) shmem[tid] += shmem[tid +  16]; __syncthreads();
        if(tid <   8) shmem[tid] += shmem[tid +   8]; __syncthreads();
        if(tid <   4) shmem[tid] += shmem[tid +   4]; __syncthreads();
        //if(tid <   2) shmem[tid] += shmem[tid +   2]; __syncthreads();

        if(tid == 0) x[j*NUM_FIBERS + (*fiber)] = shmem[0] + shmem[1] + shmem[2] + shmem[3];

        __syncthreads();
    }
}

__global__ void multiply_Aty_ECpart(
    uint32_t*  voxelEC,
    uint16_t*  orienEC,
    uint32_t*  segmentsPerBlock,
    uint32_t*  offsetPerBlock,
    float32_t* lut,
    float64_t* x,
    float64_t* y)
{
    __shared__ float64_t shmem[512];

    uint32_t bid = blockIdx.x;
    uint32_t tid = threadIdx.x;

    shmem[tid] = 0.0;

    if(tid >= NUM_SAMPLES) return;

    uint32_t offset  = offsetPerBlock[bid];
    uint32_t ncompartments = segmentsPerBlock[bid] + offset;

    //compartmentEC_t* peak = peaks + offset;
    uint32_t* voxel = voxelEC + offset;
    uint16_t* orien = orienEC + offset;

    for(int j = 0; j < NUM_ZEPPELINS; j++){        
        uint32_t offset_lut = j*NUM_ORIENTATIONS*NUM_SAMPLES + tid;

        //peak = peaks + offset;
        voxel = voxelEC + offset;
        orien = orienEC + offset;
        for(int i = offset; i < ncompartments; i++){
            //shmem[tid] =( (float64_t)tex1Dfetch(tex_lutEC, (*orient)*num_samples + offset_lut) )* y[(*voxel)*num_samples + tid];
            shmem[tid] =( (float64_t)(lut[(*orien)*NUM_SAMPLES + offset_lut] ))* y[(*voxel)*NUM_SAMPLES + tid];
            __syncthreads();

            //if(bid == 0){
            //printf("%lf\n", lut[(peak->orientation)*num_samples + lut_offset] * y[(peak->voxel)*num_samples + tid]);

            if(tid < 256) shmem[tid] += shmem[tid + 256]; __syncthreads();
            if(tid < 128) shmem[tid] += shmem[tid + 128]; __syncthreads();
            if(tid <  64) shmem[tid] += shmem[tid +  64]; __syncthreads();
            if(tid <  32) shmem[tid] += shmem[tid +  32]; __syncthreads();
            if(tid <  16) shmem[tid] += shmem[tid +  16]; __syncthreads();
            if(tid <   8) shmem[tid] += shmem[tid +   8]; __syncthreads();
            if(tid <   4) shmem[tid] += shmem[tid +   4]; __syncthreads();
            if(tid <   2) shmem[tid] += shmem[tid +   2]; __syncthreads();

            if(tid == 0) x[NUM_FIBERS*NUM_DIAMETERS + j*NUM_PEAKS + i] = shmem[0] + shmem[1];
            //}

            //peak++;
            voxel++;
            orien++;
            __syncthreads();
        }
    }
} //*/

__global__ void multiply_Aty_ISOpart(float* lut, double* x, double* y){
    __shared__ double shmem[512];

    uint bid = blockIdx.x;
    uint tid = threadIdx.x;
    uint offset = NUM_FIBERS*NUM_DIAMETERS + NUM_PEAKS*NUM_ZEPPELINS + bid;

    shmem[tid] = 0.0;

    if(tid >= NUM_SAMPLES) return;

    for(int j = 0; j < NUM_BALLS; j++){
        shmem[tid] =( (float64_t) lut[j*NUM_SAMPLES + tid] )* y[bid*NUM_SAMPLES + tid];
        //shmem[tid] =( (float64_t) tex1Dfetch(tex_lutISO, j*num_samples + tid) )* y[bid*num_samples + tid];
        __syncthreads();

        if(tid < 256) shmem[tid] += shmem[tid + 256]; __syncthreads();
        if(tid < 128) shmem[tid] += shmem[tid + 128]; __syncthreads();
        if(tid <  64) shmem[tid] += shmem[tid +  64]; __syncthreads();
        if(tid <  32) shmem[tid] += shmem[tid +  32]; __syncthreads();
        if(tid <  16) shmem[tid] += shmem[tid +  16]; __syncthreads();
        if(tid <   8) shmem[tid] += shmem[tid +   8]; __syncthreads();
        if(tid <   4) shmem[tid] += shmem[tid +   4]; __syncthreads(); 

        if(tid == 0)
            x[offset + j*NUM_VOXELS] = shmem[0] + shmem[1] + shmem[2] + shmem[3];
    }
}//*/

/*void CudaLinearOperator::multiplyByX(float64_t* x, float64_t* y){

    // Copy vector x to the GPU
    hipMemcpy(this->x, x, ncols*sizeof(double), hipMemcpyHostToDevice);

    // Multiply IC part in the GPU
    multiply_Ax_ICpart<<<nvoxels, 1024>>>(voxelIC, fiberIC, orienIC, lengthIC, segmentsPerBlockIC, offsetPerBlockIC, lutIC, this->x, this->y);

    //cudaCheckKernel();

    // Multiply EC part in the GPU
    multiply_Ax_ECpart<<<nvoxels, 512>>>(voxelEC, orienEC, segmentsPerBlockEC, offsetPerBlockEC, lutEC, this->x, this->y);

    //cudaCheckKernel();

    // Multiply ISO part in the GPU
    multiply_Ax_ISOpart<<<nvoxels, 512>>>(lutISO, this->x, this->y);

    //cudaCheckKernel();

    // Copy back result to CPU
    hipMemcpy(y, this->y, nrows*sizeof(double), hipMemcpyDeviceToHost);
}*/

/*void CudaLinearOperator::multiplyByY(float64_t* v_in, float64_t* v_out){
        
    // Copy vector y to the GPU
    //cudaCheck( hipMemset(gpu_x, 0, NUM_COLS*sizeof(float64_t)) );
    //cudaCheck( hipMemcpy(gpu_x, x, NUM_COLS*sizeof(double), hipMemcpyHostToDevice) );
    cudaCheck( hipMemcpy(y, v_in, nrows*sizeof(double), hipMemcpyHostToDevice) );

    // Multiply IC part in the GPU
    multiply_Aty_ICpart<<<nfibers, 512>>>(voxelICt, fiberICt, orienICt, lengthICt, fibersPerBlockICt, offsetPerBlockICt, lutIC, x, y);

    //cudaCheckKernel();//*/

    // Multiply EC part in the GPU
    multiply_Aty_ECpart<<<nvoxels, 512>>>(voxelEC, orienEC, segmentsPerBlockEC, offsetPerBlockEC, lutEC, x, y);

    //cudaCheckKernel();

    // Multiply ISO part in the GPU
    multiply_Aty_ISOpart<<<nvoxels, 512>>>(lutISO, x, y);

    //cudaCheckKernel();//*/

    // Copy back result to CPU
    cudaCheck( hipMemcpy(v_out, x, ncols*sizeof(double), hipMemcpyDeviceToHost) );
        
    /*printf("\n\n VECTOR X EC PART:\n");
    for(int i = NUM_FIBERS*NUM_RESFUNCIC; i < NUM_FIBERS*NUM_RESFUNCIC+20; i++)
        printf("%lf ", x[i]);
    printf("\n\n");//*/
}*/